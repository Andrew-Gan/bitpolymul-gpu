#include "hip/hip_runtime.h"
/*
Copyright (C) 2017 Ming-Shing Chen

This file is part of BitPolyMul.

BitPolyMul is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

BitPolyMul is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with BitPolyMul.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "bc.h"

#include <stdio.h>

#include "util.cuh"

#define BC_CODE_GEN

static inline
unsigned get_num_blocks( unsigned poly_len , unsigned blk_size ) {
	return poly_len/blk_size;
}


static inline
unsigned deg_si( unsigned si ) {
	return (1<<si);
}

static inline
unsigned get_si_2_pow( unsigned si , unsigned deg ) {
	unsigned si_deg = (1<<si);
	unsigned r=1;
	while( (si_deg<<r) < deg ) {
		r += 1;
	}
	return (1<<(r-1));
}

static inline
unsigned get_max_si( unsigned deg ) {
	unsigned si = 0;
	unsigned si_attempt = 1;
	uint64_t deg64 = deg;
	while( deg64 > ((1ULL)<<si_attempt) ) {
		si = si_attempt;
		si_attempt <<= 1;
	}
	return si;
}


//////////////////////////////////////////////////////////////////////


//#include <x86intrin.h>
#include <emmintrin.h>
#include <immintrin.h>

#include "bc_to_mono_gen_code.cu"
#include "bc_to_lch_gen_code.cu"


#define LOG2(X) ((unsigned) (8*sizeof (unsigned long long) - __builtin_clzll((X)) - 1))
#define MAX(x,y) (((x)>(y))?(x):(y))
#define MIN(x,y) (((x)<(y))?(x):(y))

__global__
void xor_down_d_2(u256 *poly256, bc_sto_t *poly, unsigned st, unsigned diff) {
	uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
	u256_xor(*(poly256-i-1), *(u256*)(poly+st+diff-(i*4)-4));
}

__global__
void xor_down_d_3(bc_sto_t *poly, unsigned st, unsigned diff, unsigned start) {
	uint64_t i = blockIdx.x * blockDim.x + threadIdx.x + start;
	poly[st-i-1] ^= poly[st-i-1+diff];
}

void xor_down( bc_sto_t * poly , unsigned st , unsigned len , unsigned diff )
{
#if 0
	for( unsigned i=0;i<len;i++) {
		poly[st-i-1] ^= poly[st-i-1+diff];
	}
#else
	while( ((unsigned long)(poly+st)) & 31 ) {
		poly[st-1] ^= poly[st-1+diff];
		st--;
		len--;
		if( 0 == len ) break;
	}
	u256 * poly256 = (u256*)(poly+st);
	unsigned _len = len>>2;
	uint64_t nBlock = (_len + 1023) / 1024;
	xor_down_d_2<<<nBlock, 1024>>>(poly256, poly, st, diff);
	nBlock = (len - (_len<<2) + 1023) / 1024;
	xor_down_d_3<<<nBlock, 1024>>>(poly, st, diff, _len<<2);
#endif
}

static inline
void poly_div( bc_sto_t * poly , unsigned n_terms , unsigned blk_size , unsigned si , unsigned pow )
{
	if( 0 == si ) return;
	unsigned si_degree = deg_si(si)*pow;
	unsigned deg_diff = si_degree - pow;
	unsigned deg_blk = get_num_blocks( n_terms , blk_size ) -1;
#if 1
	xor_down( poly , (deg_blk-deg_diff+1)*blk_size , (deg_blk-si_degree+1)*blk_size , deg_diff*blk_size );
#else
	for(unsigned i=deg_blk;i>=si_degree;i--) {
		for(int j=((int)blk_size)-1;j>=0;j--) {
			poly[(i-deg_diff)*blk_size+j] ^= poly[i*blk_size+j];
		}
	}
#endif
}

static inline
void represent_in_si( bc_sto_t * poly , unsigned n_terms , unsigned blk_size , unsigned si )
{
	if( 0 == si ) return;
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned degree_basic_form_si = deg_si(si);
	if( degree_basic_form_si > degree_in_blocks ) return;

#if 1
	unsigned pow = get_si_2_pow( si , degree_in_blocks );
	while( 0 < pow ) {
		for(unsigned i=0;i<n_terms;i+= blk_size*2*pow*deg_si(si) ) {
			poly_div( poly + i , blk_size*2*pow*deg_si(si) , blk_size , si , pow );
		}
		pow >>= 1;
	}
#else
	unsigned pow = get_si_2_pow( si , degree_in_blocks );
	poly_div( poly , n_terms , blk_size , si , pow );
	if( 1 < pow ) {
		represent_in_si( poly , pow*deg_si(si)*blk_size , blk_size , si );
		represent_in_si( poly + pow*deg_si(si)*blk_size , n_terms - pow*deg_si(si)*blk_size , blk_size , si );
	}
#endif
}


void _bc_to_lch( bc_sto_t * poly , unsigned n_terms , unsigned blk_size )
{
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned si = get_max_si( degree_in_blocks );
	represent_in_si( poly , n_terms , blk_size , si );

	unsigned new_blk_size = deg_si(si)*blk_size;
	_bc_to_lch( poly , n_terms , new_blk_size );
	for(unsigned i=0;i<n_terms;i+= new_blk_size ) {
		_bc_to_lch( poly + i , new_blk_size , blk_size );
	}
}


void bc_to_lch( bc_sto_t * poly , unsigned n_terms )
{
	_bc_to_lch( poly , n_terms , 1 );
}



/////////////////////////////////////

__global__
void xor_up_d_2(u256 *poly256, bc_sto_t *poly, unsigned st, unsigned diff) {
	uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
	poly256[i] = u256_xor(poly256[i], *(u256*)(poly+st+diff+(i*4)));
}

__global__
void xor_up_d_3(bc_sto_t *poly, unsigned st, unsigned diff, unsigned start) {
	uint64_t i = blockIdx.x * blockDim.x + threadIdx.x + start;
	poly[st+i] ^= poly[st+i+diff];
}

static inline
void xor_up( bc_sto_t * poly , unsigned st , unsigned len , unsigned diff )
{
#if 0
	for( unsigned i=0;i<len;i++) {
		poly[st+i] ^= poly[st+i+diff];
	}
#else
	while( ((unsigned long)(poly+st)) & 31 ) {
		poly[st] ^= poly[st+diff];
		st++;
		len--;
		if( 0 == len ) break;
	}
	u256 * poly256 = (u256*)(poly+st);
	unsigned _len = len>>2;
	uint64_t nBlock = (_len + 1023) / 1024;
	xor_up_d_2<<<nBlock, 1024>>>(poly256, poly, st, diff);
	nBlock = (len - (_len<<2) + 1023) / 1024;
	xor_up_d_3<<<nBlock, 1024>>>(poly, st, diff, _len<<2);
#endif
}


static inline
void i_poly_div( bc_sto_t * poly , unsigned n_terms , unsigned blk_size , unsigned si , unsigned pow )
{
	if( 0 == si ) return;
	unsigned si_degree = deg_si(si)*pow;
	unsigned deg_diff = si_degree - pow;
	unsigned deg_blk = get_num_blocks( n_terms , blk_size ) -1;
#if 1
	xor_up( poly , (blk_size)*(si_degree-deg_diff) , (deg_blk-si_degree+1)*blk_size , deg_diff*blk_size );
#else
	for(unsigned i=si_degree;i<=deg_blk;i++) {
		for(unsigned j=0; j<blk_size ;j++) {
			poly[(i-deg_diff)*blk_size+j] ^= poly[i*blk_size+j];
		}
	}
#endif
}

static inline
void i_represent_in_si( bc_sto_t * poly , unsigned n_terms , unsigned blk_size , unsigned si )
{
	if( 0 == si ) return;
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned degree_basic_form_si = deg_si(si);
	if( degree_basic_form_si > degree_in_blocks ) return;

	unsigned pow = 1;
	while( pow*deg_si(si) <= degree_in_blocks ) {
		for(unsigned i=0;i<n_terms;i+= blk_size*2*pow*deg_si(si) ) {
			i_poly_div( poly + i , blk_size*2*pow*deg_si(si) , blk_size , si , pow );
		}
		pow *= 2;

	}
}


void _bc_to_mono( bc_sto_t * poly , unsigned n_terms , unsigned blk_size )
{
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned si = get_max_si( degree_in_blocks );


	unsigned new_blk_size = deg_si(si)*blk_size;
	for(unsigned i=0;i<n_terms;i+= new_blk_size ) {
		_bc_to_mono( poly + i , new_blk_size , blk_size );
	}
	_bc_to_mono( poly , n_terms , new_blk_size );
	i_represent_in_si( poly , n_terms , blk_size , si );
}


void bc_to_mono( bc_sto_t * poly , unsigned n_terms )
{
	_bc_to_mono( poly , n_terms , 1 );
}





//////////////////////////////////////////////


static inline
void xor_down_128( __m128i * poly , unsigned st , unsigned len , unsigned diff )
{
#if 0
	for( unsigned i=0;i<len;i++) {
		poly[st-i-1] ^= poly[st-i-1+diff];
	}
#else
	if( ((unsigned long)(poly+st)) & 31 ) {
		poly[st-1] ^= poly[st+diff-1];
		st--;
		len--;
	}
	__m256i * poly256 = (__m256i*)(poly+st);
	unsigned _len = len>>1;
	for( unsigned i=0;i<_len;i++ ) {
		*(poly256 - i-1) ^= _mm256_loadu_si256( (__m256i*)(poly+st+diff-(i*2)-2) );
	}
	if( len&1 ) {
		poly[st-len] ^= poly[st-len+diff];
	}
#endif
}



static inline
void poly_div_128( __m128i * poly , unsigned n_terms , unsigned blk_size , unsigned si , unsigned pow )
{
	if( 0 == si ) return;
	unsigned si_degree = deg_si(si)*pow;
	unsigned deg_diff = si_degree - pow;
	unsigned deg_blk = get_num_blocks( n_terms , blk_size ) -1;

	xor_down_128( poly , (deg_blk-deg_diff+1)*blk_size , (deg_blk-si_degree+1)*blk_size , deg_diff*blk_size );
}

static inline
void represent_in_si_128( __m128i * poly , unsigned n_terms , unsigned blk_size , unsigned si )
{
	if( 0 == si ) return;
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned degree_basic_form_si = deg_si(si);
	if( degree_basic_form_si > degree_in_blocks ) return;

#if 1
	unsigned pow = get_si_2_pow( si , degree_in_blocks );
	while( 0 < pow ) {
		for(unsigned i=0;i<n_terms;i+= blk_size*2*pow*deg_si(si) ) {
			poly_div_128( poly + i , blk_size*2*pow*deg_si(si) , blk_size , si , pow );
		}
		pow >>= 1;
	}
#else
	unsigned pow = get_si_2_pow( si , degree_in_blocks );
	poly_div( poly , n_terms , blk_size , si , pow );
	if( 1 < pow ) {
		represent_in_si( poly , pow*deg_si(si)*blk_size , blk_size , si );
		represent_in_si( poly + pow*deg_si(si)*blk_size , n_terms - pow*deg_si(si)*blk_size , blk_size , si );
	}
#endif
}


void _bc_to_lch_128( __m128i * poly , unsigned n_terms , unsigned blk_size )
{
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned si = get_max_si( degree_in_blocks );
	represent_in_si_128( poly , n_terms , blk_size , si );

	unsigned new_blk_size = deg_si(si)*blk_size;
	_bc_to_lch_128( poly , n_terms , new_blk_size );
	for(unsigned i=0;i<n_terms;i+= new_blk_size ) {
		_bc_to_lch_128( poly + i , new_blk_size , blk_size );
	}
}


void bc_to_lch_128( bc_sto_t * poly , unsigned n_terms )
{
	_bc_to_lch_128( (__m128i*) poly , n_terms , 1 );
}


///////////////////////////////////


static inline
void xor_up_128( __m128i * poly , unsigned st , unsigned len , unsigned diff )
{
#if 0
	for( unsigned i=0;i<len;i++) {
		poly[st+i] ^= poly[st+i+diff];
	}
#else
	if( ((unsigned long)(poly+st)) & 31 ) {
		poly[st] ^= poly[st+diff];
		st++;
		len--;
	}
	__m256i * poly256 = (__m256i*)(poly+st);
	unsigned _len = len>>1;
	for( unsigned i=0;i<_len;i++ ) {
		poly256[i] ^= _mm256_loadu_si256( (__m256i*)(poly+st+diff+(i*2)) );
	}
	if( len&1 ) {
		poly[st+len-1] ^= poly[st+len-1+diff];
	}
#endif
}


static inline
void i_poly_div_128( __m128i * poly , unsigned n_terms , unsigned blk_size , unsigned si , unsigned pow )
{
	if( 0 == si ) return;
	unsigned si_degree = deg_si(si)*pow;
	unsigned deg_diff = si_degree - pow;
	unsigned deg_blk = get_num_blocks( n_terms , blk_size ) -1;

#if 1
	xor_up_128( poly , blk_size*(si_degree-deg_diff) , (deg_blk-si_degree+1)*blk_size , deg_diff*blk_size );
#else
	for(unsigned i=si_degree;i<=deg_blk;i++) {
		for(unsigned j=0; j<blk_size ;j++) {
			poly[(i-deg_diff)*blk_size+j] ^= poly[i*blk_size+j];
		}
	}
#endif
}

static inline
void i_represent_in_si_128( __m128i * poly , unsigned n_terms , unsigned blk_size , unsigned si )
{
	if( 0 == si ) return;
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned degree_basic_form_si = deg_si(si);
	if( degree_basic_form_si > degree_in_blocks ) return;

	unsigned pow = 1;
	while( pow*deg_si(si) <= degree_in_blocks ) {
		for(unsigned i=0;i<n_terms;i+= blk_size*2*pow*deg_si(si) ) {
			i_poly_div_128( poly + i , blk_size*2*pow*deg_si(si) , blk_size , si , pow );
		}
		pow *= 2;
	}
}


void _bc_to_mono_128( __m128i * poly , unsigned n_terms , unsigned blk_size )
{

//printf("ibc: %d/%d\n", n_terms , blk_size );

	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;

//printf("deg: %d\n", degree_in_blocks);
	unsigned si = get_max_si( degree_in_blocks );
//printf("si: %d\n",si);

	unsigned new_blk_size = deg_si(si)*blk_size;
//printf("new blksize: %d\n", new_blk_size);
	for(unsigned i=0;i<n_terms;i+= new_blk_size ) {
		_bc_to_mono_128( poly + i , new_blk_size , blk_size );
	}
	_bc_to_mono_128( poly , n_terms , new_blk_size );
	i_represent_in_si_128( poly , n_terms , blk_size , si );
}


void bc_to_mono_128( bc_sto_t * poly , unsigned n_terms )
{

	_bc_to_mono_128( (__m128i*)poly , n_terms , 1 );
}





//////////////////////////////////////////////

// Beispiel:
// __xor_down_256<<<dim3(1, 1), src_idx - dest_idx>>>(...);
__global__
void __xor_down_256( u256 * poly , unsigned dest_idx , unsigned src_idx, unsigned len, unsigned unit ) {
	uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t pSize = gridDim.x * blockDim.x;

	if (i >= src_idx - dest_idx) return;
	poly += blockIdx.y * unit;

	for (uint64_t d = (dest_idx+len-pSize), s = (src_idx+len-pSize); d >= dest_idx && s >= src_idx; d -= pSize, s -= pSize) {
		poly[d+i] = u256_xor(poly[d+i], poly[s+i]);
	}
}

static inline
void xor_down_256( u256 * poly , unsigned st , unsigned len , unsigned diff )
{
	unsigned dest_st = st - len;
	unsigned src_st = st - len + diff;

	uint64_t nBlock = (diff + 1023) / 1024;
	__xor_down_256<<<nBlock, 1024>>>(poly, dest_st, src_st, len, 1);
//	for( unsigned i=0;i<len;i++) {
//		poly[st-i-1] ^= poly[st-i-1+diff];
//	}
}

static inline
void __xor_down_256_2( u256 * poly , unsigned len , unsigned l_st, unsigned num, unsigned unit ) {
	dim3 nBlock((len - l_st + 1023) / 1024, num);
	__xor_down_256<<<nBlock, 1024>>>(poly, l_st, len, len, unit);
}

// Beispiel:
// __xor_up_256<<<1, src_idx - dest_idx>>>(...);
__global__
void __xor_up_256( u256 * poly , unsigned dest_idx , unsigned src_idx , unsigned len, unsigned unit ) {
	uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t pSize = gridDim.x * blockDim.x;

	if (i >= src_idx - dest_idx) return;
	poly += blockIdx.y * unit;

	for (uint64_t d = dest_idx, s = src_idx; (d+i) < (dest_idx+len) && (s+i) < (src_idx+len); d += pSize, s += pSize) {
		poly[d+i] = u256_xor(poly[d+i], poly[s+i]);
	}
}

static inline
void xor_up_256( u256 * poly , unsigned st , unsigned len , unsigned diff )
{
	uint64_t nBlock = (diff + 1023) / 1024;
	__xor_up_256<<<nBlock, 1024>>>( poly , st , st + diff, len, 1 );
//	for( unsigned i=0;i<len;i++) {
//		poly[st+i] ^= poly[st+i+diff];
//	}
}

static inline
void __xor_up_256_2( u256 * poly , unsigned len , unsigned l_st, unsigned num, unsigned unit ){
	dim3 nBlock((len - l_st + 1023) / 1024, num);
	__xor_up_256<<<nBlock, 1024>>>(poly, l_st, len, len, unit);
//	for( unsigned i=0;i<len;i++) poly[l_st+i] ^= poly[len+i];
}



//////////////////////////////////////////////////////////////////////////



static inline
void poly_div_256( u256 * poly , unsigned n_terms , unsigned blk_size , unsigned si , unsigned pow )
{
	if( 0 == si ) return;
	unsigned si_degree = deg_si(si)*pow;
	unsigned deg_diff = si_degree - pow;
	unsigned deg_blk = get_num_blocks( n_terms , blk_size ) -1;

	xor_down_256( poly , (deg_blk-deg_diff+1)*blk_size , (deg_blk-si_degree+1)*blk_size , deg_diff*blk_size );
}

static inline
void represent_in_si_256( u256 * poly , unsigned n_terms , unsigned blk_size , unsigned si )
{
	if( 0 == si ) return;
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned degree_basic_form_si = deg_si(si);
	if( degree_basic_form_si > degree_in_blocks ) return;

#if 1
	unsigned pow = get_si_2_pow( si , degree_in_blocks );
	while( 0 < pow ) {
		for(unsigned i=0;i<n_terms;i+= blk_size*2*pow*deg_si(si) ) {
			poly_div_256( poly + i , blk_size*2*pow*deg_si(si) , blk_size , si , pow );
		}
		pow >>= 1;
	}
#else
	unsigned pow = get_si_2_pow( si , degree_in_blocks );
	poly_div( poly , n_terms , blk_size , si , pow );
	if( 1 < pow ) {
		represent_in_si( poly , pow*deg_si(si)*blk_size , blk_size , si );
		represent_in_si( poly + pow*deg_si(si)*blk_size , n_terms - pow*deg_si(si)*blk_size , blk_size , si );
	}
#endif
}


void _bc_to_lch_256( u256 * poly , unsigned n_terms , unsigned blk_size )
{

	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned si = get_max_si( degree_in_blocks );
	represent_in_si_256( poly , n_terms , blk_size , si );

	unsigned new_blk_size = deg_si(si)*blk_size;
	_bc_to_lch_256( poly , n_terms , new_blk_size );
	for(unsigned i=0;i<n_terms;i+= new_blk_size ) {
		_bc_to_lch_256( poly + i , new_blk_size , blk_size );
	}
}


void bc_to_lch_256( bc_sto_t * poly , unsigned n_terms )
{
	_bc_to_lch_256( (u256*) poly , n_terms , 1 );
}


///////////////////////////////////



static inline
void i_poly_div_256( u256 * poly , unsigned n_terms , unsigned blk_size , unsigned si , unsigned pow )
{
	if( 0 == si ) return;
	unsigned si_degree = deg_si(si)*pow;
	unsigned deg_diff = si_degree - pow;
	unsigned deg_blk = get_num_blocks( n_terms , blk_size ) -1;

	xor_up_256( poly , blk_size*(si_degree-deg_diff) , (deg_blk-si_degree+1)*blk_size , deg_diff*blk_size );
}

static inline
void i_represent_in_si_256( u256 * poly , unsigned n_terms , unsigned blk_size , unsigned si )
{
	if( 0 == si ) return;
	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned degree_basic_form_si = deg_si(si);
	if( degree_basic_form_si > degree_in_blocks ) return;

	unsigned pow = 1;
	while( pow*deg_si(si) <= degree_in_blocks ) {
		for(unsigned i=0;i<n_terms;i+= blk_size*2*pow*deg_si(si) ) {
			i_poly_div_256( poly + i , blk_size*2*pow*deg_si(si) , blk_size , si , pow );
		}
		pow *= 2;
	}
}


void _bc_to_mono_256( u256 * poly , unsigned n_terms , unsigned blk_size )
{

	unsigned num_blocks = get_num_blocks( n_terms , blk_size );
	if( 2 >= num_blocks ) return;
	unsigned degree_in_blocks = num_blocks - 1;
	unsigned si = get_max_si( degree_in_blocks );

	unsigned new_blk_size = deg_si(si)*blk_size;
	for(unsigned i=0;i<n_terms;i+= new_blk_size ) {
		_bc_to_mono_256( poly + i , new_blk_size , blk_size );
	}
	_bc_to_mono_256( poly , n_terms , new_blk_size );
	i_represent_in_si_256( poly , n_terms , blk_size , si );
}


void bc_to_mono_256( bc_sto_t * poly , unsigned n_terms )
{

	_bc_to_mono_256( (u256*)poly , n_terms , 1 );
}





///////////////////////////////////////////////

#define NDEBUG
#include "assert.h"



//#include "byte_inline_func.h"

__device__
u256 _mm256_alignr_255bit_zerohigh( u256 zerohigh , u256 low )
{
	// __m256i l_shr_15 = _mm256_srli_epi16( low , 15 );
	// __m256i r_1 = _mm256_permute2x128_si256( l_shr_15 , zerohigh , 0x21 );
	// return _mm256_srli_si256( r_1 , 14 );

	u256 l_shr_15 = srli<uint16_t>(low, 15);
	u256 r_1 = permute2x128(l_shr_15, zerohigh, 0x21);
	return srli<uint8_t>(r_1, 14);
}

__device__
u256 _mm256_alignr_254bit_zerohigh( u256 zerohigh , u256 low )
{
	// __m256i l_shr_14 = _mm256_srli_epi16( low , 14 );
	// __m256i r_2 = _mm256_permute2x128_si256( l_shr_14 , zerohigh , 0x21 );
	// return _mm256_srli_si256( r_2 , 14 );

	u256 l_shr_14 = srli<uint16_t>(low, 14);
	u256 r_2 = permute2x128(l_shr_14, zerohigh, 0x21);
	return srli<uint8_t>(r_2, 14);
}

__device__
u256 _mm256_alignr_252bit_zerohigh( u256 zerohigh , u256 low )
{
	// __m256i l_shr_12 = _mm256_srli_epi16( low , 12 );
	// __m256i r_4 = _mm256_permute2x128_si256( l_shr_12 , zerohigh , 0x21 );
	// return _mm256_srli_si256( r_4 , 14 );

	u256 l_shr_12 = srli<uint16_t>(low, 12);
	u256 r_4 = permute2x128(l_shr_12, zerohigh, 0x21);
	return srli<uint8_t>(r_4, 14);
}

__device__
u256 _mm256_alignr_255bit( u256 high , u256 low )
{
	// __m256i l_shr_15 = _mm256_srli_epi16( low , 15 );
	// __m256i h_shr_15 = _mm256_srli_epi16( high , 15 );
	// __m256i h_shl_1 = _mm256_slli_epi16( high , 1 );
	// __m256i r = h_shl_1^_mm256_slli_si256( h_shr_15 , 2 );

	// __m256i r_1 = _mm256_permute2x128_si256( l_shr_15 , h_shr_15 , 0x21 );
	// r ^= _mm256_srli_si256( r_1 , 14 );
	// return r;

	u256 l_shr_15 = srli<uint16_t>(low, 15);
	u256 h_shr_15 = srli<uint16_t>(high, 15);
	u256 h_shl_1 = slli<uint16_t>(high, 1);
	u256 r = u256_xor(h_shl_1, slli<uint8_t>(h_shr_15, 2));
	return r;
}

__device__
u256 _mm256_alignr_254bit( u256 high , u256 low )
{
	// __m256i l_shr_14 = _mm256_srli_epi16( low , 14 );
	// __m256i h_shr_14 = _mm256_srli_epi16( high , 14 );
	// __m256i h_shl_2 = _mm256_slli_epi16( high , 2 );
	// __m256i r = h_shl_2^_mm256_slli_si256( h_shr_14 , 2 );

	// __m256i r_2 = _mm256_permute2x128_si256( l_shr_14 , h_shr_14 , 0x21 );
	// r ^= _mm256_srli_si256( r_2 , 14 );
	// return r;

	u256 l_shr_14 = srli<uint16_t>(low, 14);
	u256 h_shr_14 = srli<uint16_t>(high, 14);
	u256 h_shl_2 = slli<uint16_t>(high, 2);
	u256 r = u256_xor(h_shl_2, slli<uint8_t>(h_shr_14, 2));

	u256 r_2 = permute2x128(l_shr_14, h_shr_14 , 0x21);
	r = u256_xor(r, srli<uint8_t>(r_2, 14));
	return r;
}

__device__
u256 _mm256_alignr_252bit( u256 high , u256 low )
{
	// __m256i l_shr_12 = _mm256_srli_epi16( low , 12 );
	// __m256i h_shr_12 = _mm256_srli_epi16( high , 12 );
	// __m256i h_shl_4 = _mm256_slli_epi16( high , 4 );
	// __m256i r = h_shl_4^_mm256_slli_si256( h_shr_12 , 2 );

	// __m256i r_4 = _mm256_permute2x128_si256( l_shr_12 , h_shr_12 , 0x21 );
	// r ^= _mm256_srli_si256( r_4 , 14 );
	// return r;

	u256 l_shr_12 = srli<uint16_t>(low, 12);
	u256 h_shr_12 = srli<uint16_t>(high, 12);
	u256 h_shl_4 = srli<uint16_t>(high, 4);
	u256 r = u256_xor(h_shl_4, slli<uint8_t>(h_shr_12, 2));
	

	u256 r_4 = permute2x128(l_shr_12, h_shr_12 , 0x21);
	r = u256_xor(r, srli<uint8_t>(r_4, 14));
	return r;
}

__device__
u256 _mm256_alignr_31byte( u256 high , u256 low )
{
	// __m256i l0 = _mm256_permute2x128_si256( low , high , 0x21 );
	// return _mm256_alignr_epi8( high , l0 , 15 );

	u256 l0 = permute2x128(low, high, 0x21);
	return alignr(high, l0, 15);
	
}

__device__
u256 _mm256_alignr_30byte( u256 high , u256 low )
{
	// __m256i l0 = _mm256_permute2x128_si256( low , high , 0x21 );
	// return _mm256_alignr_epi8( high , l0 , 14 );

	u256 l0 = permute2x128(low, high, 0x21);
	return alignr(high, l0 , 14);
}

__device__
u256 _mm256_alignr_28byte( u256 high , u256 low )
{
	// __m256i l0 = _mm256_permute2x128_si256( low , high , 0x21 );
	// return _mm256_alignr_epi8( high , l0 , 12 );

	u256 l0 = permute2x128(low, high, 0x21);
	return alignr(high, l0 , 12);
}

__device__
u256 _mm256_alignr_24byte( u256 high , u256 low )
{
	// __m256i l0 = _mm256_permute2x128_si256( low , high , 0x21 );
	// return _mm256_alignr_epi8( high , l0 , 8 );

	u256 l0 = permute2x128(low, high, 0x21);
	return alignr(high, l0 , 8);
}

__device__
u256 _mm256_alignr_16byte( u256 high , u256 low )
{
	// return _mm256_permute2x128_si256( low , high , 0x21 );

	return permute2x128(low, high , 0x21);
}

__device__
u256 (*_sh_op[8]) (u256 h, u256 l) = {
	_mm256_alignr_255bit, _mm256_alignr_254bit, _mm256_alignr_252bit, _mm256_alignr_31byte, _mm256_alignr_30byte, _mm256_alignr_28byte, _mm256_alignr_24byte, _mm256_alignr_16byte
};

__device__
u256 (*_sh_op_zerohigh[8]) (u256 h, u256 l) = {
	_mm256_alignr_255bit_zerohigh , _mm256_alignr_254bit_zerohigh , _mm256_alignr_252bit_zerohigh , _mm256_alignr_31byte, _mm256_alignr_30byte, _mm256_alignr_28byte, _mm256_alignr_24byte, _mm256_alignr_16byte
};

__global__
void __sh_xor_down_a(u256 *poly256, unsigned unit, unsigned _op, u256 zero) {
	poly256 += (blockIdx.x * blockDim.x + threadIdx.x) * unit;
	poly256[(unit>>1)] = u256_xor(poly256[(unit>>1)], _sh_op_zerohigh[_op](zero,poly256[unit-1]));
}

__global__
void __sh_xor_down_b(u256 *poly256, unsigned unit, unsigned _op) {
	uint64_t i = blockIdx.y;
	poly256 += (blockIdx.x * blockDim.x + threadIdx.x) * unit;
	poly256[(unit>>1)-1-i] = u256_xor(poly256[(unit>>1)-1-i], _sh_op[_op](poly256[unit-1-i] , poly256[unit-2-i]));
}

__global__
void __sh_xor_down_c(u256 *poly256, unsigned unit, unsigned _op, u256 zero) {
	poly256 += (blockIdx.x * blockDim.x + threadIdx.x) * unit;
	poly256[0] = u256_xor(poly256[0], _sh_op[_op](poly256[unit>>1],zero));
}

static inline
void __sh_xor_down( u256* poly256 , unsigned unit , unsigned _op , u256 zero, unsigned num )
{
	__sh_xor_down_a<<<(num+1023)/1024, 1024>>>(poly256, unit, _op, zero);

	dim3 nBlock((num+1023)/1024, (unit>>1)-1);
	if (nBlock.x > 0 && nBlock.y > 0)
		__sh_xor_down_b<<<nBlock, 1024>>>(poly256, unit, _op);

	__sh_xor_down_c<<<(num+1023)/1024, 1024>>>(poly256, unit, _op, zero);
}


static
void varsub_x256( u256* poly256 , unsigned n_256 )
{
	if( 1 >= n_256 ) return;
	unsigned log_n = __builtin_ctz( n_256 );
	u256 zero;
	// __m256i zero = _mm256_setzero_si256();

	while( log_n > 8 ) {
		unsigned unit = 1<<log_n;
		unsigned num = n_256/unit;
		unsigned unit_2 = unit>>1;
		__xor_down_256_2( poly256 , unit_2 , (1<<(log_n-9)), num, unit );
		log_n--;
	}

	for(unsigned i=log_n; i>0 ; i--) {
		unsigned unit = (1<<i);
		unsigned num = n_256 / unit;
		__sh_xor_down( poly256 , unit , i-1 , zero, num );
	}
}

__global__
void __sh_xor_up_a(u256 *poly256, unsigned unit, unsigned _op, u256 zero) {
	poly256 += (blockIdx.x * blockDim.x + threadIdx.x) * unit;
	poly256[0] = u256_xor(poly256[0], _sh_op[_op](poly256[unit>>1],zero));
}

__global__
void __sh_xor_up_b(u256 *poly256, unsigned unit, unsigned _op) {
	uint64_t i = blockIdx.y;
	poly256 += (blockIdx.x * blockDim.x + threadIdx.x) * unit;
	poly256[i+1] = u256_xor(poly256[i+1], _sh_op[_op]( poly256[(unit>>1)+i+1] , poly256[(unit>>1)+i] ));
}

__global__
void __sh_xor_up_c(u256 *poly256, unsigned unit, unsigned _op, u256 zero) {
	poly256 += (blockIdx.x * blockDim.x + threadIdx.x) * unit;
	poly256[(unit>>1)] = u256_xor(poly256[(unit>>1)], _sh_op_zerohigh[_op](zero,poly256[unit-1]));
}

static inline
void __sh_xor_up( u256* poly256 , unsigned unit , unsigned _op , u256 zero, unsigned num )
{
	__sh_xor_up_a<<<(num+1023)/1024, 1024>>>(poly256, unit, _op, zero);

	dim3 nBlock((num+1023)/1024, (unit>>1)-1);
	if (nBlock.x > 0 && nBlock.y > 0)
		__sh_xor_up_b<<<nBlock, 1024>>>(poly256, unit, _op);

	__sh_xor_up_c<<<(num+1023)/1024, 1024>>>(poly256, unit, _op, zero);
}

static
void i_varsub_x256( u256* poly256 , unsigned n_256 )
{
	if( 1 >= n_256 ) return;
	unsigned log_n = __builtin_ctz( n_256 );
	u256 zero;
	// __m256i zero = _mm256_setzero_si256();

	unsigned _log_n = (log_n>8)? 8 : log_n;
	for(unsigned i=1; i<=_log_n ; i++) {
		unsigned unit = (1<<i);
		unsigned num = n_256 / unit;
		__sh_xor_up( poly256 , unit , i-1 , zero, num );
	}

	for(unsigned i=9;i<=log_n ; i++ ) {
		unsigned unit = 1<<i;
		unsigned num = n_256/unit;
		unsigned unit_2 = unit>>1;
		__xor_up_256_2( poly256 , unit_2 , (1<<(i-9)), num, unit );
	}
}


void bc_to_lch_2_unit256( bc_sto_t * poly , unsigned n_terms )
{
	assert( 0 == ( n_terms&(n_terms-1) ) );
	assert( 4 <= n_terms );

	u256 * poly256 = (u256*) poly;
	unsigned n_256 = n_terms>>2;

	// varsub_x256( poly256 , n_256 );

#ifdef BC_CODE_GEN
	int logn = LOG2(n_256);
	// bc_to_lch_256_30_12(poly256,logn);

	// for(int i=0;i<(1<<(MAX(0,logn-19)));++i){
	//     bc_to_lch_256_19_17(poly256+i*(1<<19),MIN(19,logn));
	// }

	struct timespec start, end;

	clock_gettime(CLOCK_MONOTONIC, &start);
	// for(int i=0;i<(1<<(MAX(0,logn-16)));++i){
	// 	bc_to_lch_256_16(poly256+i*(1<<16), MIN(16,logn));
	// }
	bc_to_lch_256_16(poly256, MIN(16,logn));
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end);

	float loopDuration = (end.tv_sec - start.tv_sec) * 1000;
	loopDuration += (end.tv_nsec - start.tv_nsec) / 1000000.0;
	printf("Overall: %.2f ms\n", loopDuration);

#else
	_bc_to_lch_256( poly256 , n_256 , 1 );
#endif
}


void bc_to_mono_2_unit256( bc_sto_t * poly , unsigned n_terms )
{
	assert( 0 == ( n_terms&(n_terms-1) ) );
	assert( 4 <= n_terms );

	u256 * poly256 = (u256*) poly;
	unsigned n_256 = n_terms>>2;

#ifdef BC_CODE_GEN
        int logn = LOG2(n_256);
        for(int i=0;i<(1<<(MAX(0,logn-16)));++i){
	    	bc_to_mono_256_16(poly256+i*(1<<16), MIN(16,logn));
        }
        for(int i=0;i<(1<<(MAX(0,logn-19)));++i){
            bc_to_mono_256_19_17(poly256+i*(1<<19),MIN(19,logn));
        }
        bc_to_mono_256_30_20(poly256,logn);
#else
	_bc_to_mono_256( poly256 , n_256 , 1 );
#endif
	i_varsub_x256( poly256 , n_256 );
}


